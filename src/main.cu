#include "hip/hip_runtime.h"
#include "head.h"

//字母说明，dec为分解，signal为信号，Len为长度，filter为滤波器，
 //EX为延拓，CON为卷积，DSam为下采样，D为下，sam为采样。L为低频信号，H为高频信号
 //ref为重构，
__global__ void gpu()
{
    //std::cout << "hello gpu!" << std::endl;
    printf("hello gpu\n");
}

int main()
{
    Matrix max(4);
    max.Matrix_Init();
    max.creatMatrix();
    max.Print_matrix();
    hipSetDevice(0);
    gpu<<<1,20>>>();

    hipDeviceReset();

    return 0;
}